#include "hip/hip_runtime.h"
#include <iostream> 
#include <hip/hip_runtime.h> 
#include <stdio.h>
#include <time.h>
#include <ctime>

using namespace std;

//#define TEST_MODE

#ifdef TEST_MODE
#define MATRIX_SIZE 2048

#define MATRIX_SIZE_X MATRIX_SIZE // 512, 1024, 2048, 4096, 8192, 16384, 32768
#define MATRIX_SIZE_Y MATRIX_SIZE
#else
#define MATRIX_SIZE_X 32768 // 512, 1024, 2048, 4096, 8192, 16384, 32768
#define MATRIX_SIZE_Y 8192
#endif

#define BlockSize 32

void checkCUDAStatus(hipError_t cudaStatus);

void matrixSortWithCuda(short int A[][MATRIX_SIZE_X], short int B[][MATRIX_SIZE_X * 2]);

//Compear matrixs
void compearMtx(short int A[][MATRIX_SIZE_X * 2], short int B[][MATRIX_SIZE_X * 2], string say, bool debugMode = false)
{
	printf("\nCompear array: %s : ", say);
	int buf = 0;
	for (int i = 0; i < MATRIX_SIZE_Y / 2; i++) {
		for (int j = 0; j < MATRIX_SIZE_X * 2; j++) {
			if (!(A[i][j] == B[i][j]))
			{
				printf("Matrix is not equal\n");
				printf("\t\t %d %d \t\t\n \t\t %d != %d \t\t\n", i, j, A[i][j], B[i][j]);
				system("pause");
			}
			else if(debugMode) printf("\n\n suc %d %d : i j %d %d\n", A[i][j], B[i][j], i, j);
		}
	}
	printf("Matrix equal");
}

//CPU sort
void sortcpumatx(short int A[][MATRIX_SIZE_X], short int B[][MATRIX_SIZE_X * 2])
{
	clock_t begin, end;
	begin = clock();

	for (int i = 0; i < MATRIX_SIZE_Y; i++)
	{
		for (int j = 0; j < MATRIX_SIZE_X; j++)
		{
			B[i / 2][j * 2 + i % 2] = A[i][j];
		}
	}
	end = clock();
	printf("CPU time: %lf seconds\n", (double)(end - begin) / CLOCKS_PER_SEC);
}

int main() {
	srand(time(0));

	int(*matrixA1)[MATRIX_SIZE_X] = new int[MATRIX_SIZE_Y][MATRIX_SIZE_X];
	int(*matrixB1)[MATRIX_SIZE_X * 2] = new int[MATRIX_SIZE_Y / 2][MATRIX_SIZE_X * 2];

	short int(*matrixA)[MATRIX_SIZE_X] = reinterpret_cast<short int(*)[MATRIX_SIZE_X]>(matrixA1);
	short int(*matrixB)[MATRIX_SIZE_X * 2] = reinterpret_cast<short int(*)[MATRIX_SIZE_X * 2]>(matrixB1);

	int(*matrixC1)[MATRIX_SIZE_X * 2] = new int[MATRIX_SIZE_Y / 2][MATRIX_SIZE_X * 2];
	short int(*matrixC)[MATRIX_SIZE_X * 2] = reinterpret_cast<short int(*)[MATRIX_SIZE_X * 2]>(matrixC1);

	for (int i = 0; i < MATRIX_SIZE_Y; i++) {
		for (int j = 0; j < MATRIX_SIZE_X; j++) {
			matrixA[i][j] = rand() % 1024;
		}
	}
	for (int i = 0; i < MATRIX_SIZE_Y / 2; i++) {
		for (int j = 0; j < MATRIX_SIZE_X * 2; j++) {
			matrixB[i][j] = 0;
			matrixC[i][j] = 0;
		}
	}

	matrixSortWithCuda(matrixA, matrixB);
	sortcpumatx(matrixA, matrixC);
	compearMtx(matrixB, matrixC, "myGPU and myCPU");

	printf("\n\n");
	system("pause");

	delete[] matrixA;
	delete[] matrixB;
	delete[] matrixC;
}

__global__ void mysort(short int *a, short int *b)
{
	int column = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	if (column >= MATRIX_SIZE_X / 2 || row >= MATRIX_SIZE_Y / 2) {
		return;
	}

	int mat_i = threadIdx.x + blockIdx.x * blockDim.x + (threadIdx.y * 2 + blockIdx.y * blockDim.y * 2) * (blockDim.x * gridDim.x);
	int mat_i2 = threadIdx.x + blockIdx.x * blockDim.x + ((threadIdx.y * 2) + 1 + blockIdx.y * blockDim.y * 2) * (blockDim.x * gridDim.x);

	int str1 = *(int*)(&a[mat_i * 2]);
	int str2 = *(int*)(&a[mat_i2 * 2]);

	short int a1 = ((short int*)&str1)[0];
	short int a2 = ((short int*)&str1)[1];
	short int a3 = ((short int*)&str2)[0];
	short int a4 = ((short int*)&str2)[1];

	long long int res;
	((short int*)&res)[0] = a1;
	((short int*)&res)[1] = a3;
	((short int*)&res)[2] = a2;
	((short int*)&res)[3] = a4;

	int mat_i3 = threadIdx.x * 4 + blockIdx.x * blockDim.x * 4 + (threadIdx.y + blockIdx.y * blockDim.y) * (blockDim.x * gridDim.x * 4);
	*(long long*)(&b[mat_i3]) = res;
}

void matrixSortWithCuda(short int A[][MATRIX_SIZE_X], short int B[][MATRIX_SIZE_X * 2])
{
	short int *dev_a, *dev_b;
	clock_t begin, end;
	hipError_t cudaStatus;
	hipEvent_t start;
	hipEvent_t stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);


	cudaStatus = hipMalloc((void**)&dev_a, ((MATRIX_SIZE_Y)*(MATRIX_SIZE_X)) * sizeof(short int));
	checkCUDAStatus(cudaStatus);
	cudaStatus = hipMalloc((void**)&dev_b, ((MATRIX_SIZE_X * 2)*(MATRIX_SIZE_Y / 2)) * sizeof(short int));
	checkCUDAStatus(cudaStatus);

	cudaStatus = hipMemcpy(dev_a, A, ((MATRIX_SIZE_X * MATRIX_SIZE_Y)) * sizeof(short int), hipMemcpyHostToDevice);
	checkCUDAStatus(cudaStatus);

	dim3 dimBlock(BlockSize, BlockSize);
	dim3 dimGrid((MATRIX_SIZE_X / 2) / dimBlock.x, (MATRIX_SIZE_Y / 2) / dimBlock.y);

	hipEventRecord(start);

	mysort << < dimGrid, dimBlock >> > (dev_a, dev_b);

	hipEventRecord(stop);
	hipEventSynchronize(stop);

	cudaStatus = hipGetLastError();
	checkCUDAStatus(cudaStatus);

	cudaStatus = hipMemcpy(B, dev_b, ((MATRIX_SIZE_X * 2 * MATRIX_SIZE_Y / 2)) * sizeof(short int), hipMemcpyDeviceToHost);
	checkCUDAStatus(cudaStatus);

	float time;
	hipEventElapsedTime(&time, start, stop);

	printf("CUDA time: %f seconds\n", time / 1000);

	hipFree(dev_a);
	hipFree(dev_b);
}

void checkCUDAStatus(hipError_t cudaStatus) {
	if (cudaStatus != hipSuccess) {
		printf("CUDA return error code: %d\n", cudaStatus);
		system("pause");
		exit(-1);
	}
}