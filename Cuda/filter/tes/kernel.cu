#include "hip/hip_runtime.h"
// test version 1 for 5 lab

#include <iostream> 
#include <hip/hip_runtime.h> 
#include <stdio.h>
#include <time.h>
#include <ctime>
#include <fstream>

using namespace std;

struct BitmapPGM
{
	char type[3];
	char coment[18] = "# Created by Tim\n";
	int max_size;
	int size_x;
	int size_y;
	unsigned char *bitmap;

	int chanel;

	BitmapPGM() {}
	BitmapPGM(BitmapPGM *clone, unsigned char *bmp)
	{
		strcpy(type, clone->type);
		max_size = clone->max_size;
		size_x = clone->size_x;
		size_y = clone->size_y;
		chanel = clone->chanel;
		bitmap = bmp;

	}
};

struct RGBBmp
{
	BitmapPGM r;
	BitmapPGM g;
	BitmapPGM b;
};
struct RGBCC
{
	unsigned char r;
	unsigned char g;
	unsigned char b;
};
void openPGM(const char* file, BitmapPGM *bmp) {
	FILE *fp = NULL;
	fopen_s(&fp, file, "rb");

	const unsigned int PGMHeaderSize = 255;
	char header[PGMHeaderSize];

	fgets(header, PGMHeaderSize, fp);

	if (header[0] == 'P' && header[1] == '5')
	{
		bmp->type[0] = 'P';
		bmp->type[1] = '5';
		bmp->type[2] = '\n';
		bmp->chanel = 1;
	}
	else
	{
		bmp->type[0] = 'P';
		bmp->type[1] = '6';
		bmp->type[2] = '\n';
		bmp->chanel = 3;
	}

	for (int i = 0; i < 3;)
	{
		fgets(header, PGMHeaderSize, fp);
		if (header[0] == '#') continue;
		if (header[0] == '\n') continue;
		if (i == 0) i += sscanf_s(header, "%u %u %u", &bmp->size_x, &bmp->size_y, &bmp->max_size);
		else if (i == 1) i += sscanf_s(header, "%u %u", &bmp->size_x, &bmp->size_y);
		else if (i == 2) i += sscanf_s(header, "%u", &bmp->max_size);
	}

	bmp->bitmap = (unsigned char *)malloc(sizeof(unsigned char) * bmp->size_x * bmp->size_y * bmp->chanel);
	fread(bmp->bitmap, sizeof(unsigned char), bmp->size_x * bmp->size_y * bmp->chanel, fp);

	fclose(fp);
}
bool savePGM(const char *file, BitmapPGM bmp) {

	std::fstream fh(file, std::fstream::out | std::fstream::binary);

	fh << bmp.type;
	fh << bmp.coment;


	fh << bmp.size_x << '\n' << bmp.size_y << "\n" << bmp.max_size << std::endl;

	for (unsigned int i = 0; (i < (bmp.size_x * bmp.size_y * bmp.chanel)) && fh.good(); ++i)
	{
		fh << bmp.bitmap[i];
	}

	fh.flush();

	if (fh.bad())
	{
		cout << "Writing data failed." << endl;
		return false;
	}

	fh.close();

	return true;
}
BitmapPGM filterCPU(BitmapPGM i_bmp)
{
	unsigned char *res = (unsigned char *)malloc(sizeof(unsigned char) * i_bmp.size_x * i_bmp.size_y);
	const unsigned long long start = clock();
	for (int y = 0; y < i_bmp.size_y; y++)
	{
		for (int x = 0; x < i_bmp.size_x; x++)
		{
			int pos = x + y * i_bmp.size_x;
			int sum = 0;
			for (int i = -1; i < 2; i++)
			{
				for (int j = -1; j < 2; j++)
				{
					if (i == 0 && j == 0)
					{
						sum += i_bmp.bitmap[pos] * (-8);
					}
					else if (x + j >= 0 && x + j < i_bmp.size_x && y + i >= 0 && y + i < i_bmp.size_y)
					{
						sum += i_bmp.bitmap[j + x + (y + i) * i_bmp.size_x];
					}
				}
			}
			if (sum >= 0 && sum <= 255) res[pos] = sum;
			else if (sum < 0) res[pos] = 0;
			else res[pos] = 255;
		}
	}

	const unsigned long long end = clock();
	cout << "CPU filter work: " << ((end - start) / (double)CLOCKS_PER_SEC) << " sec" << endl;

	return BitmapPGM(&i_bmp, res);
}
void checkCUDAStatus(hipError_t cudaStatus) {
	if (cudaStatus != hipSuccess) {
		printf("CUDA return error code: %d\n", cudaStatus);
		system("pause");
		exit(-1);
	}
}

RGBBmp raspilRGB(BitmapPGM bmp)
{
	RGBBmp cBmp;
	unsigned char *R = (unsigned char *)malloc(sizeof(unsigned char) * bmp.size_x * bmp.size_y);
	unsigned char *G = (unsigned char *)malloc(sizeof(unsigned char) * bmp.size_x * bmp.size_y);
	unsigned char *B = (unsigned char *)malloc(sizeof(unsigned char) * bmp.size_x * bmp.size_y);

	for (int i = 0; i < bmp.size_x * bmp.size_y; i++)
	{
		R[i] = bmp.bitmap[i * 3];
		G[i] = bmp.bitmap[i * 3 + 1];
		B[i] = bmp.bitmap[i * 3 + 2];
	}

	cBmp.r = BitmapPGM(&bmp, R);
	cBmp.g = BitmapPGM(&bmp, G);
	cBmp.b = BitmapPGM(&bmp, B);

	cBmp.r.chanel = 1;
	cBmp.g.chanel = 1;
	cBmp.b.chanel = 1;

	return cBmp;
}

BitmapPGM mergingRGB(RGBBmp cBmp)
{
	unsigned char *arr = (unsigned char *)malloc(sizeof(unsigned char) * cBmp.r.size_x * cBmp.r.size_y * 3);

	for (int i = 0; i < cBmp.r.size_x * cBmp.r.size_y; i++)
	{
		arr[i * 3] = cBmp.r.bitmap[i];
		arr[i * 3 + 1] = cBmp.g.bitmap[i];
		arr[i * 3 + 2] = cBmp.b.bitmap[i];
	}

	BitmapPGM bmp(&cBmp.r, arr);
	bmp.chanel = 3;
	return bmp;
}

// threadds = x1024 y0 block = x: w / 1024 + 1 y: 4

__global__ void filterGPU4(const int h, const int w, const RGBCC *in, RGBCC *out, int res_pitch, int out_pitch)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x + 1;
	int idy = threadIdx.y + blockIdx.y * blockDim.y + 1;

	//if (idx == w - 1 && idy < h) out[idy * out_pitch + w - 1] = 255;

	int idx2 = idx - 1;

	if (blockIdx.y == 0)
	{
		if (idx2 == 0)
		{
			int sum = in[0].r * (-8) + in[1].r + in[0 + res_pitch].r + in[1 + res_pitch].r;
			int sum1 = in[0].g * (-8) + in[1].g + in[0 + res_pitch].g + in[1 + res_pitch].g;
			int sum2 = in[0].b * (-8) + in[1].b + in[0 + res_pitch].b + in[1 + res_pitch].b;

			//printf("\n0 0: %d + %d + %d + %d = %d", in[0] * (-8), in[1], in[0 + res_pitch], in[1 + res_pitch], sum);

			if ((sum) >= 0 && (sum) <= 255) out[0].r = (unsigned char)(sum);
			else if ((sum) < 0) out[0].r = (unsigned char)0;
			else if ((sum) > 255) out[0].r = (unsigned char)255;

			if ((sum) >= 0 && (sum) <= 255) out[0].g = (unsigned char)(sum1);
			else if ((sum) < 0) out[0].g = (unsigned char)0;
			else if ((sum) > 255) out[0].g = (unsigned char)255;
			
			if ((sum) >= 0 && (sum) <= 255) out[0].b = (unsigned char)(sum2);
			else if ((sum) < 0) out[0].b = (unsigned char)0;
			else if ((sum) > 255) out[0].b = (unsigned char)255;

		}
		else if (idx2 == w - 1)
		{
			int sum = in[w - 1].r * (-8) + in[w - 2].r + in[w - 2 + res_pitch].r + in[w - 1 + res_pitch].r;
			//printf("\n0 w: %d + %d + %d + %d = %d\n", in[w - 1] * (-8), in[w - 2], in[w - 2 + res_pitch], in[w - 1 + res_pitch], sum);
			if ((sum) >= 0 && (sum) <= 255) out[w - 1].r = (unsigned char)(sum);
			else if ((sum) < 0) out[w - 1].r = (unsigned char)0;
			else if ((sum) > 255) out[w - 1].r = (unsigned char)255;

			sum = in[w - 1].g * (-8) + in[w - 2].g + in[w - 2 + res_pitch].g + in[w - 1 + res_pitch].g;
			//printf("\n0 w: %d + %d + %d + %d = %d\n", in[w - 1] * (-8), in[w - 2], in[w - 2 + res_pitch], in[w - 1 + res_pitch], sum);
			if ((sum) >= 0 && (sum) <= 255) out[w - 1].g = (unsigned char)(sum);
			else if ((sum) < 0) out[w - 1].g = (unsigned char)0;
			else if ((sum) > 255) out[w - 1].g = (unsigned char)255;

			sum = in[w - 1].b * (-8) + in[w - 2].b + in[w - 2 + res_pitch].b + in[w - 1 + res_pitch].b;
			//printf("\n0 w: %d + %d + %d + %d = %d\n", in[w - 1] * (-8), in[w - 2], in[w - 2 + res_pitch], in[w - 1 + res_pitch], sum);
			if ((sum) >= 0 && (sum) <= 255) out[w - 1].b = (unsigned char)(sum);
			else if ((sum) < 0) out[w - 1].b = (unsigned char)0;
			else if ((sum) > 255) out[w - 1].b = (unsigned char)255;
		}else
		if (idx2 < w)
		{
			int sum = in[idx2].r * (-8) + in[idx2 - 1].r + in[idx2 + 1].r + in[idx2 + res_pitch - 1].r + in[idx2 + res_pitch + 1].r + in[idx2 + res_pitch].r;
			if ((sum) >= 0 && (sum) <= 255) out[idx2].r = (unsigned char)(sum);
			else if ((sum) < 0) out[idx2].r = (unsigned char)0;
			else if ((sum) > 255) out[idx2].r = (unsigned char)255;

			sum = in[idx2].g * (-8) + in[idx2 - 1].g + in[idx2 + 1].g + in[idx2 + res_pitch - 1].g + in[idx2 + res_pitch + 1].g + in[idx2 + res_pitch].g;
			if ((sum) >= 0 && (sum) <= 255) out[idx2].g = (unsigned char)(sum);
			else if ((sum) < 0) out[idx2].g = (unsigned char)0;
			else if ((sum) > 255) out[idx2].g = (unsigned char)255;

			sum = in[idx2].b * (-8) + in[idx2 - 1].b + in[idx2 + 1].b + in[idx2 + res_pitch - 1].b + in[idx2 + res_pitch + 1].b + in[idx2 + res_pitch].b;
			if ((sum) >= 0 && (sum) <= 255) out[idx2].b = (unsigned char)(sum);
			else if ((sum) < 0) out[idx2].b = (unsigned char)0;
			else if ((sum) > 255) out[idx2].b = (unsigned char)255;
		}
	}
	if (blockIdx.y == 1)
	{
		if (idx2 == 0)
		{
			return;
		}else
		if (idx2 == h - 1)
		{
			int sum = in[(h - 1)*res_pitch].r * (-8) + in[(h - 1)*res_pitch + 1].r + in[(h - 2)*res_pitch].r + in[(h - 2)*res_pitch + 1].r;
			if ((sum) >= 0 && (sum) <= 255) out[(h - 1)*out_pitch].r = (unsigned char)(sum);
			else if ((sum) < 0) out[(h - 1)*out_pitch].r = (unsigned char)0;
			else if ((sum) > 255) out[(h - 1)*out_pitch].r = (unsigned char)255;

			sum = in[(h - 1)*res_pitch].g * (-8) + in[(h - 1)*res_pitch + 1].g + in[(h - 2)*res_pitch].g + in[(h - 2)*res_pitch + 1].g;
			if ((sum) >= 0 && (sum) <= 255) out[(h - 1)*out_pitch].g = (unsigned char)(sum);
			else if ((sum) < 0) out[(h - 1)*out_pitch].g = (unsigned char)0;
			else if ((sum) > 255) out[(h - 1)*out_pitch].g = (unsigned char)255;

			sum = in[(h - 1)*res_pitch].b * (-8) + in[(h - 1)*res_pitch + 1].b + in[(h - 2)*res_pitch].b + in[(h - 2)*res_pitch + 1].b;
			if ((sum) >= 0 && (sum) <= 255) out[(h - 1)*out_pitch].b = (unsigned char)(sum);
			else if ((sum) < 0) out[(h - 1)*out_pitch].b = (unsigned char)0;
			else if ((sum) > 255) out[(h - 1)*out_pitch].b = (unsigned char)255;
		} else 
			if (idx2 >= h)
			{
				int sum = in[idx2 * res_pitch].r * (-8) + in[idx2 * res_pitch + 1].r + in[(idx2 - 1) * res_pitch].r + in[(idx2 - 1) * res_pitch + 1].r + in[(idx + 1) * res_pitch].r + in[(idx + 1) * res_pitch + 1].r;
				if ((sum) >= 0 && (sum) <= 255) out[idx2 * res_pitch].r = (unsigned char)(sum);
				else if ((sum) < 0) out[idx2 * res_pitch].r = (unsigned char)0;
				else if ((sum) > 255) out[idx2 * res_pitch].r = (unsigned char)255;

				sum = in[idx2 * res_pitch].g * (-8) + in[idx2 * res_pitch + 1].g + in[(idx2 - 1) * res_pitch].g + in[(idx2 - 1) * res_pitch + 1].g + in[(idx + 1) * res_pitch].g + in[(idx + 1) * res_pitch + 1].g;
				if ((sum) >= 0 && (sum) <= 255) out[idx2 * res_pitch].g = (unsigned char)(sum);
				else if ((sum) < 0) out[idx2 * res_pitch].g = (unsigned char)0;
				else if ((sum) > 255) out[idx2 * res_pitch].g = (unsigned char)255;

				sum = in[idx2 * res_pitch].b * (-8) + in[idx2 * res_pitch + 1].b + in[(idx2 - 1) * res_pitch].b + in[(idx2 - 1) * res_pitch + 1].b + in[(idx + 1) * res_pitch].b + in[(idx + 1) * res_pitch + 1].b;
				if ((sum) >= 0 && (sum) <= 255) out[idx2 * res_pitch].b = (unsigned char)(sum);
				else if ((sum) < 0) out[idx2 * res_pitch].b = (unsigned char)0;
				else if ((sum) > 255) out[idx2 * res_pitch].b = (unsigned char)255;
			}
	}
	if (blockIdx.y == 2)
	{
		if (idx2 >= h) return;
		if (idx2 == 0)
		{
			return;
		}
		else if (idx2 == h - 1)
		{
			int sum = in[(h - 1) * res_pitch + w - 1].r * (-8) + in[(h - 2) * res_pitch + w - 1].r + in[(h - 2) * res_pitch + w - 1 - 1].r + in[(h - 1) * res_pitch + w - 1 - 1].r;
			if ((sum) >= 0 && (sum) <= 255) out[(h - 1) * out_pitch + w - 1].r = (unsigned char)(sum);
			else if ((sum) < 0) out[(h - 1) * out_pitch + w - 1].r = (unsigned char)0;
			else if ((sum) > 255) out[(h - 1) * out_pitch + w - 1].r = (unsigned char)255;

			sum = in[(h - 1) * res_pitch + w - 1].g * (-8) + in[(h - 2) * res_pitch + w - 1].g + in[(h - 2) * res_pitch + w - 1 - 1].g + in[(h - 1) * res_pitch + w - 1 - 1].g;
			if ((sum) >= 0 && (sum) <= 255) out[(h - 1) * out_pitch + w - 1].g = (unsigned char)(sum);
			else if ((sum) < 0) out[(h - 1) * out_pitch + w - 1].g = (unsigned char)0;
			else if ((sum) > 255) out[(h - 1) * out_pitch + w - 1].g = (unsigned char)255;

			sum = in[(h - 1) * res_pitch + w - 1].b * (-8) + in[(h - 2) * res_pitch + w - 1].b + in[(h - 2) * res_pitch + w - 1 - 1].b + in[(h - 1) * res_pitch + w - 1 - 1].b;
			if ((sum) >= 0 && (sum) <= 255) out[(h - 1) * out_pitch + w - 1].b = (unsigned char)(sum);
			else if ((sum) < 0) out[(h - 1) * out_pitch + w - 1].b = (unsigned char)0;
			else if ((sum) > 255) out[(h - 1) * out_pitch + w - 1].b = (unsigned char)255;
		}
		else
		{
			int sum = in[idx2 * res_pitch + w - 1].r * (-8) + in[idx2 * res_pitch + w - 1 - 1].r + in[(idx2 - 1) * res_pitch + w - 1].r + in[(idx2 - 1) * res_pitch + w - 1 - 1].r
				+ in[(idx2 + 1) * res_pitch + w - 1].r + in[(idx2 + 1) * res_pitch + w - 1 - 1].r;
			if ((sum) >= 0 && (sum) <= 255) out[idx2 * res_pitch + w - 1].r = (unsigned char)(sum);
			else if ((sum) < 0) out[idx2 * res_pitch + w - 1].r = (unsigned char)0;
			else if ((sum) > 255) out[idx2 * res_pitch + w - 1].r = (unsigned char)255;

			sum = in[idx2 * res_pitch + w - 1].g * (-8) + in[idx2 * res_pitch + w - 1 - 1].g + in[(idx2 - 1) * res_pitch + w - 1].g + in[(idx2 - 1) * res_pitch + w - 1 - 1].g
				+ in[(idx2 + 1) * res_pitch + w - 1].g + in[(idx2 + 1) * res_pitch + w - 1 - 1].g;
			if ((sum) >= 0 && (sum) <= 255) out[idx2 * res_pitch + w - 1].g = (unsigned char)(sum);
			else if ((sum) < 0) out[idx2 * res_pitch + w - 1].g = (unsigned char)0;
			else if ((sum) > 255) out[idx2 * res_pitch + w - 1].g = (unsigned char)255;

			sum = in[idx2 * res_pitch + w - 1].b * (-8) + in[idx2 * res_pitch + w - 1 - 1].b + in[(idx2 - 1) * res_pitch + w - 1].b + in[(idx2 - 1) * res_pitch + w - 1 - 1].b
				+ in[(idx2 + 1) * res_pitch + w - 1].b + in[(idx2 + 1) * res_pitch + w - 1 - 1].b;
			if ((sum) >= 0 && (sum) <= 255) out[idx2 * res_pitch + w - 1].b = (unsigned char)(sum);
			else if ((sum) < 0) out[idx2 * res_pitch + w - 1].b = (unsigned char)0;
			else if ((sum) > 255) out[idx2 * res_pitch + w - 1].b = (unsigned char)255;
		}
	}
	if (blockIdx.y == 3)
	{
		if (idx2 >= w) return;
		if (idx2 == 0)
		{
			return;
		}
		else if (idx2 == w - 1)
		{
			return;
		}
		else
		{
			int sum = in[(h - 1) * res_pitch + idx2].r * (-8) + in[(h - 1) * res_pitch + idx2 - 1].r + in[(h - 1) * res_pitch + idx2 + 1].r
				+ in[(h - 1) * res_pitch + idx2 - res_pitch - 1].r + in[(h - 1) * res_pitch + idx2 - res_pitch].r + in[(h - 1) * res_pitch + idx2 - res_pitch + 1].r;
			if ((sum) >= 0 && (sum) <= 255) out[(h - 1) * res_pitch + idx2].r = (unsigned char)(sum);
			else if ((sum) < 0) out[(h - 1) * res_pitch + idx2].r = (unsigned char)0;
			else if ((sum) > 255) out[(h - 1) * res_pitch + idx2].r = (unsigned char)255;

			sum = in[(h - 1) * res_pitch + idx2].g * (-8) + in[(h - 1) * res_pitch + idx2 - 1].g + in[(h - 1) * res_pitch + idx2 + 1].g
				+ in[(h - 1) * res_pitch + idx2 - res_pitch - 1].g + in[(h - 1) * res_pitch + idx2 - res_pitch].g + in[(h - 1) * res_pitch + idx2 - res_pitch + 1].g;
			if ((sum) >= 0 && (sum) <= 255) out[(h - 1) * res_pitch + idx2].g = (unsigned char)(sum);
			else if ((sum) < 0) out[(h - 1) * res_pitch + idx2].g = (unsigned char)0;
			else if ((sum) > 255) out[(h - 1) * res_pitch + idx2].g = (unsigned char)255;

			sum = in[(h - 1) * res_pitch + idx2].b * (-8) + in[(h - 1) * res_pitch + idx2 - 1].b + in[(h - 1) * res_pitch + idx2 + 1].b
				+ in[(h - 1) * res_pitch + idx2 - res_pitch - 1].b + in[(h - 1) * res_pitch + idx2 - res_pitch].b + in[(h - 1) * res_pitch + idx2 - res_pitch + 1].b;
			if ((sum) >= 0 && (sum) <= 255) out[(h - 1) * res_pitch + idx2].b = (unsigned char)(sum);
			else if ((sum) < 0) out[(h - 1) * res_pitch + idx2].b = (unsigned char)0;
			else if ((sum) > 255) out[(h - 1) * res_pitch + idx2].b = (unsigned char)255;
		}
	}

	if (idy < h - 1 && idx < w - 1)
	{
		int a[9] = {};
		for (int z2 = 0; z2 < 3; ++z2)
		{
			for (int z1 = 0; z1 < 3; ++z1)
			{
				a[3 * z2 + z1] = in[idx + idy * res_pitch + (z2 - 1) * res_pitch + (z1 - 1)].r;// idx + idy * w + z2 * w + z1
			}
		}

		int sum = a[0] + a[1] + a[2] + a[3] + (a[4] * (-8)) + a[5] + a[6] + a[7] + a[8];

		if ((sum) >= 0 && (sum) <= 255) out[idy * out_pitch + idx].r = (unsigned char)(sum);
		else if ((sum) < 0) out[idy * out_pitch + idx].r = (unsigned char)0;
		else if ((sum) > 255) out[idy * out_pitch + idx].r = (unsigned char)255;

		for (int z2 = 0; z2 < 3; ++z2)
		{
			for (int z1 = 0; z1 < 3; ++z1)
			{
				a[3 * z2 + z1] = in[idx + idy * res_pitch + (z2 - 1) * res_pitch + (z1 - 1)].g;// idx + idy * w + z2 * w + z1
			}
		}

		sum = a[0] + a[1] + a[2] + a[3] + (a[4] * (-8)) + a[5] + a[6] + a[7] + a[8];

		if ((sum) >= 0 && (sum) <= 255) out[idy * out_pitch + idx].g = (unsigned char)(sum);
		else if ((sum) < 0) out[idy * out_pitch + idx].g = (unsigned char)0;
		else if ((sum) > 255) out[idy * out_pitch + idx].g = (unsigned char)255;

		for (int z2 = 0; z2 < 3; ++z2)
		{
			for (int z1 = 0; z1 < 3; ++z1)
			{
				a[3 * z2 + z1] = in[idx + idy * res_pitch + (z2 - 1) * res_pitch + (z1 - 1)].b;// idx + idy * w + z2 * w + z1
			}
		}

		sum = a[0] + a[1] + a[2] + a[3] + (a[4] * (-8)) + a[5] + a[6] + a[7] + a[8];

		if ((sum) >= 0 && (sum) <= 255) out[idy * out_pitch + idx].b = (unsigned char)(sum);
		else if ((sum) < 0) out[idy * out_pitch + idx].b = (unsigned char)0;
		else if ((sum) > 255) out[idy * out_pitch + idx].b = (unsigned char)255;
	}
}

BitmapPGM filterOnCuda2(BitmapPGM i_bmp)
{
	unsigned char *res = (unsigned char *)malloc(sizeof(unsigned char) * i_bmp.size_x * i_bmp.size_y * i_bmp.chanel);

	unsigned char* inputCuda;
	unsigned char* resCuda;
	size_t inputPitch;
	size_t resPitch;

	checkCUDAStatus(hipMallocPitch(&inputCuda, &inputPitch, i_bmp.size_x * 3, i_bmp.size_y));
	checkCUDAStatus(hipMallocPitch(&resCuda, &resPitch, i_bmp.size_x * 3, i_bmp.size_y));

	hipEvent_t begin, end;
	hipEventCreate(&begin);
	hipEventCreate(&end);

	hipEventRecord(begin);
	checkCUDAStatus(hipMemcpy2D(inputCuda, inputPitch, i_bmp.bitmap, i_bmp.size_x * 3, i_bmp.size_x * 3 * sizeof(unsigned char), i_bmp.size_y, hipMemcpyHostToDevice));

	int dimGrid_x = 0;
	int dimGrid_y = 0;

	if (i_bmp.size_x % 32 == 0) dimGrid_x = i_bmp.size_x / 32;
	else dimGrid_x = i_bmp.size_x / 32 + 1;
	if (i_bmp.size_y % 32 == 0) dimGrid_y = i_bmp.size_y / 16;
	else dimGrid_y = i_bmp.size_y / 32 + 1;

	dim3 dimBlock(32, 32);
	dim3 dimGrid(dimGrid_x, dimGrid_y);

	filterGPU4 << < dimGrid, dimBlock >> > (i_bmp.size_y, i_bmp.size_x, (RGBCC*)inputCuda, (RGBCC*)resCuda, (int)inputPitch / 3, (int)resPitch / 3);

	checkCUDAStatus(hipMemcpy2D(res, i_bmp.size_x * 3, resCuda, resPitch, i_bmp.size_x * 3 * sizeof(unsigned char), i_bmp.size_y, hipMemcpyDeviceToHost));

	hipEventRecord(end);
	hipDeviceSynchronize();

	float resTime = 0;
	hipEventElapsedTime(&resTime, begin, end);

	printf("CUDA time: %f seconds\n", resTime / 1000);

	return BitmapPGM(&i_bmp, res);
}

void compearBitmap(BitmapPGM bmp1, BitmapPGM bmp2)
{
	cout << "Compear bitmap: ";
	for (int j = 0; j < bmp1.size_y; j++)
	{
		for (int i = 0; i < bmp1.size_x * bmp1.chanel; i++)
		{
			if (bmp1.bitmap[i + j * bmp1.size_x] != bmp2.bitmap[i + j * bmp1.size_x])
			{
				if (bmp1.bitmap[i + j * bmp1.size_x] - bmp2.bitmap[i + j * bmp1.size_x] > 2
					|| bmp2.bitmap[i + j * bmp1.size_x] - bmp1.bitmap[i + j * bmp1.size_x] > 2)
				{
					cout << "Error in pos: " << j << " " << i << "\tbmp1: " << (int)bmp1.bitmap[i + j * bmp1.size_x] << " bmp2: " << (int)bmp2.bitmap[i + j * bmp1.size_x] << endl;
					system("pause");
				}
			}
		}
	}
	cout << "Matrix equal" << endl;
}

void main()
{
	BitmapPGM bmp;
	openPGM("test5.ppm", &bmp);

	if (bmp.chanel == 3)
	{
		RGBBmp cBmp = raspilRGB(bmp);

		RGBBmp cAns;
		cAns.r = filterCPU(cBmp.r);
		cAns.g = filterCPU(cBmp.g);
		cAns.b = filterCPU(cBmp.b);

		BitmapPGM ans = mergingRGB(cAns);

		BitmapPGM ans2 = filterOnCuda2(bmp);

		compearBitmap(ans, ans2);

		savePGM("result.ppm", ans2);
	}
	else
	{
		BitmapPGM ans = filterCPU(bmp);
		BitmapPGM ans2 = filterOnCuda2(bmp);

		compearBitmap(ans, ans2);

		savePGM("save1.pgm", ans);
	}
	system("pause");
}